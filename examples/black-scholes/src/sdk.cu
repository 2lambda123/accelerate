
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include "sdk-kernel.cu"

#define cutilSafeCallNoSync(err)     __cudaSafeCallNoSync(err, __FILE__, __LINE__)

inline void __cudaSafeCallNoSync( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : cudaSafeCallNoSync() Runtime API error : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}


extern "C"
void blackscholes
(
    float               *h_CallResult,
    float               *h_PutResult,
    float               *h_StockPrice,
    float               *h_OptionStrike,
    float               *h_OptionYears,
    const float         riskfree,
    const float         volatility,
    const int           opt_n
)
{
    const float opt_sz = opt_n * sizeof(float);
    float * d_CallResult;
    float * d_PutResult;
    float * d_StockPrice;
    float * d_OptionStrike;
    float * d_OptionYears;

    /*
     * Allocate device memory
     */
    cutilSafeCallNoSync( hipMalloc((void **)&d_CallResult,   opt_sz) );
    cutilSafeCallNoSync( hipMalloc((void **)&d_PutResult,    opt_sz) );
    cutilSafeCallNoSync( hipMalloc((void **)&d_StockPrice,   opt_sz) );
    cutilSafeCallNoSync( hipMalloc((void **)&d_OptionStrike, opt_sz) );
    cutilSafeCallNoSync( hipMalloc((void **)&d_OptionYears,  opt_sz) );

    /*
     * Copy options data to GPU
     */
    cutilSafeCallNoSync( hipMemcpy(d_StockPrice,   h_StockPrice,   opt_sz, hipMemcpyHostToDevice) );
    cutilSafeCallNoSync( hipMemcpy(d_OptionStrike, h_OptionStrike, opt_sz, hipMemcpyHostToDevice) );
    cutilSafeCallNoSync( hipMemcpy(d_OptionYears,  h_OptionYears,  opt_sz, hipMemcpyHostToDevice) );

    /*
     * Execute
     */
    BlackScholesGPU<<<480, 128>>>(
        d_CallResult,
        d_PutResult,
        d_StockPrice,
        d_OptionStrike,
        d_OptionYears,
        riskfree,
        volatility,
        opt_n
    );
    cutilSafeCallNoSync( hipDeviceSynchronize() );

    /*
     * Copy result back to host
     */
    cutilSafeCallNoSync( hipMemcpy(h_CallResult, d_CallResult, opt_sz, hipMemcpyDeviceToHost) );
    cutilSafeCallNoSync( hipMemcpy(h_PutResult,  d_PutResult,  opt_sz, hipMemcpyDeviceToHost) );

    /*
     * Release device memory
     */
    cutilSafeCallNoSync( hipFree(d_OptionYears)  );
    cutilSafeCallNoSync( hipFree(d_OptionStrike) );
    cutilSafeCallNoSync( hipFree(d_StockPrice)   );
    cutilSafeCallNoSync( hipFree(d_PutResult)    );
    cutilSafeCallNoSync( hipFree(d_CallResult)   );
}

